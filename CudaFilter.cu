

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>


#define B (128*128)
#define THREADS_PER_BLOCK 128
int N = 128,
    M = 128;
int patch_size = 7;


struct timespec start, finish, t0, t1;

__device__ void patch(double *Image, int i, int j,double *patch_array, int patch_size, int N, int M);
void G(double *G_array,int patch_size);
double* addNoise(double* Image, int imSize);
__global__ void nonLocalMeans(double *newImage,double *Image, int N, int M,double *patches,int patch_size, double *G_array);
__device__ double denoise(double *Image, double *patches, int patch_size, int N, int M, int id,double *G_array);
__global__ void CudaPatch(int patch_size,double *Image, int N, int M, double *patches);


double* readFile(char* filename, int N, int M){

    double* A = (double *)malloc(N *M  * sizeof(double));

    FILE *f = fopen(filename, "r");

    for(int i = 0; i < N * M; i++) fscanf(f, "%lf %*c", &A[i]);

    fclose(f);
    return A;
}

void writeFile(double *A, char* filename, int N, int M){

    FILE *f = fopen(filename, "w");

    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < M - 1; j++)
        {
            fprintf(f, "%lf,", A[i*N + j]);
        }
        fprintf(f, "%lf", A[i*N + M-1]);
        fprintf(f, "\n");
    }

    fclose(f);

}


int main(){

    


    double *Image =readFile("cat.txt", N, M);

    


    printf("adding noise...\n");
    Image = addNoise( Image, N*N);   //image with noise

    writeFile(Image, "noise.txt" , N, M);
    printf("noise added\n");
    
    
    int squers =  patch_size*patch_size;
    

    clock_gettime(CLOCK_REALTIME, &start);

    double *newImage = (double *)malloc(N*M * sizeof(double));
    double *G_array = (double *)malloc(patch_size*patch_size * sizeof(double));
    G(G_array, patch_size);
    


    double *d_Image, *d_newImage, *d_patches, *d_G_array;
    hipMalloc((void**)&d_Image, N*M * sizeof(double));
    hipMalloc((void**)&d_newImage, N*M * sizeof(double));
    hipMalloc((void**)&d_patches, N*M * squers * sizeof(double));
    hipMalloc((void**)&d_G_array, squers * sizeof(double));

    hipMemcpy(d_Image,  Image,  N*M * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_G_array, G_array, squers * sizeof(double), hipMemcpyHostToDevice);
    
    CudaPatch<<<B/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(patch_size, d_Image, N, M, d_patches);
    nonLocalMeans<<<B/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_newImage, d_Image, N, M, d_patches, patch_size, d_G_array);
    hipDeviceSynchronize();

    clock_gettime(CLOCK_REALTIME, &finish);


    hipMemcpy(newImage, d_newImage, N*M * sizeof(double), hipMemcpyDeviceToHost);
    writeFile(newImage, "denoise.txt" , N, M);
    double duration = ((finish.tv_sec - start.tv_sec) * 1000000 + (finish.tv_nsec - start.tv_nsec) / 1000) / 1000000.0;
    printf("Duration: %f from %d , %d \n", duration,N,patch_size);

    double *differences  = (double *)malloc(N*M*sizeof(double));
    for(int i=0; i<N*M; i++) differences [i]=fabs( -newImage[i] + Image[i]);
    writeFile(differences , "diference.txt" , N, M);
    return 0;

}
__global__ void CudaPatch(int patch_size,double *Image, int N, int M, double *patches){
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    patch(Image, blockIdx.x,threadIdx.x,(patches + patch_size*patch_size*id), patch_size,N,M);
}
__global__ void nonLocalMeans(double *newImage,double *Image, int N, int M,double *patches,int patch_size, double *G_array){

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    newImage[id] = denoise(Image, patches,  patch_size,  N,  M,  id,G_array);

}

__device__ void patch(double *Image, int i, int j,double *patch_array, int patch_size, int N, int M){
    int p=0, o=0;
    for(int u=i-patch_size/2; u<=i+patch_size/2; u++){
        for(int v=j-patch_size/2; v<=j+patch_size/2; v++){
            if(u>=0 && v>=0 && u<N && v<M) patch_array[p++] = Image[u*N +v];
            else patch_array[p++] = Image[abs(v)*M +abs(u)];
        }
    }
}

__device__ double denoise(double *Image, double *patches, int patch_size, int N, int M, int id,double *G_array){
    
    int squers = patch_size*patch_size;
    int pointer = id*squers;
    double sigma = 0.05;
    double z=0;
    double patch_Distances = 0, newPixel=0;
    
    for(int u=0; u<N*M; u++){
        for(int v=0; v<squers; v++){
            //if(patches[pointer + v] == -1 || patches[u*squers +v] == -1) continue;
            patch_Distances += (patches[pointer + v] - patches[u*squers +v])*(patches[pointer + v] - patches[u*squers +v])*G_array[v];
        }
        double w = exp(-patch_Distances/pow(sigma,2));
        z += w;
        newPixel += w*Image[u];
        patch_Distances = 0;
    }

    return newPixel/z;
}

void G(double *G_array,int patch_size){

    int *array = (int *)malloc(patch_size*patch_size*2 * sizeof(int));

    double sigma = 5/3.0;
    int p=0;
    for(int i=-patch_size/2; i<=patch_size/2; i++){
        for(int j=-patch_size/2; j<=patch_size/2; j++){
            array[p++] = i;
            array[p++] = j;
        }
    }
    
    p=0;
    double z=0;
    for( ; p<2*patch_size*patch_size; ){
        z+=exp(-(pow(array[p++],2) + pow(array[p++],2))/(2*sigma*sigma));
    }

    p=0;
    for( int i=0; p<2*patch_size*patch_size; i++){
        G_array[i] = (1/(sqrt(2*M_PI)*sigma))*exp(-(pow(array[p++],2) + pow(array[p++],2))/(2*sigma*sigma));
    }
    
}

double* addNoise(double* Image, int imSize){

    double *noise = (double *)malloc(imSize * sizeof(double));
    double sigma = 2,value, effect;

    for(int i = 0; i < imSize; i++)
    {
        value    = ((double)( rand() ) / RAND_MAX*20 - 10);
        effect   = (1 / (sigma*sqrt(2*M_PI)))*exp((-value*value) / (2*sigma*sigma)) - 0.1;
        noise[i] = (0.5*effect + 1) * Image[i];
    }
    
    return noise;
}